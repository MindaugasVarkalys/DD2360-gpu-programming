
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <stdio.h>

#define NUM_PARTICLES 10000
#define NUM_ITERATIONS 10000

#define TPB 256
#define N (NUM_PARTICLES/TPB + 1)

struct Particle {
    float position_x;
    float position_y;
    float position_z;

    float velocity_x;
    float velocity_y;
    float velocity_z;
};

__global__ void simulate(Particle* particles) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    Particle* p = &particles[id];

    hiprandState state;
    hiprand_init(id, id, 0, &state);
    p->velocity_x += hiprand_uniform(&state);
    p->velocity_y += hiprand_uniform(&state);
    p->velocity_z += hiprand_uniform(&state);

    p->position_x += p->velocity_x;
    p->position_y += p->velocity_y;
    p->position_z += p->velocity_z;
}

int main()
{
    Particle *particles = new Particle[NUM_PARTICLES];
    Particle *d_particles = new Particle[NUM_PARTICLES];

    //cudaMalloc(&d_particles, sizeof(Particle) * NUM_PARTICLES);
    hipHostMalloc(&d_particles, sizeof(Particle) * NUM_PARTICLES, hipHostMallocDefault);

    for (int i = 0; i < NUM_ITERATIONS; i++) {
        hipMemcpy(d_particles, particles, sizeof(Particle) * NUM_PARTICLES, hipMemcpyHostToDevice);
        simulate<<<N, TPB>>>(d_particles);
        hipMemcpy(particles, d_particles, sizeof(Particle) * NUM_PARTICLES, hipMemcpyDeviceToHost);
    }
}