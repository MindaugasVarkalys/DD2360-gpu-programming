
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <stdio.h>

#define NUM_PARTICLES 256
#define NUM_ITERATIONS 10000
#define NUM_STREAMS 2

#define TPB 256
#define N (NUM_PARTICLES/TPB + 1)

struct Particle {
    float position_x;
    float position_y;
    float position_z;

    float velocity_x;
    float velocity_y;
    float velocity_z;
};

__global__ void simulate(Particle* particles, int offset) {
    int id = offset + threadIdx.x + blockIdx.x * blockDim.x;
    Particle* p = &particles[id];

    hiprandState state;
    hiprand_init(id, id, 0, &state);

    for (int i = 0; i < NUM_ITERATIONS; i++) {
        p->velocity_x += hiprand_uniform(&state);
        p->velocity_y += hiprand_uniform(&state);
        p->velocity_z += hiprand_uniform(&state);

        p->position_x += p->velocity_x;
        p->position_y += p->velocity_y;
        p->position_z += p->velocity_z;
    }
}

int main()
{
    Particle *particles = new Particle[NUM_PARTICLES];
    Particle *d_particles = new Particle[NUM_PARTICLES];
    hipHostMalloc(&d_particles, sizeof(Particle) * NUM_PARTICLES, hipHostMallocDefault);

    int streamSize = NUM_PARTICLES / NUM_STREAMS;
    int streamBytes = streamSize * sizeof(Particle);
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_particles[offset], &particles[offset], streamBytes, hipMemcpyHostToDevice, streams[i]);
        simulate<<<N, TPB>>>(d_particles, offset);
        hipMemcpyAsync(&particles[offset], &d_particles[offset], streamBytes, hipMemcpyDeviceToHost, streams[i]);
    }

    hipDeviceSynchronize();

    for (int i = 0; i < 10; i++) {
        printf("%f %f %f\n", particles[i].position_x, particles[i].position_y, particles[i].position_z);
    }
}